#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <numeric>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

__device__ volatile unsigned d_succ = 0;
__device__ volatile unsigned d_fail = 0;


__global__ void allot_request(int *centre,int *facility,int *capacity,int *fac_ids,int *req_id,int *req_cen,int *req_fac,int *req_start,int *req_slots,int i,int R){

  unsigned int id = threadIdx.x;
  __shared__ unsigned int access_buffer[1024];
  __shared__ unsigned int req_id_buffer[1024];
  __shared__ unsigned int temp_buffer[1024];
  __shared__ unsigned int temp2_buffer[1024];
  __shared__ int size;
  size = 1;

  if(id < R){
    unsigned int uid = req_cen[id] * 100 + req_fac[id];
    access_buffer[id] = uid;
    req_id_buffer[id] = req_id[id];

    if (threadIdx.x == 0 ){
        int l1,l2,k,h1,h2,j;
          for(size=1; size < R; size=size*2)
          {
            l1=0;
            k=0;
            while( l1+size < R)
            {
              h1=l1+size-1;
              l2=h1+1;
              h2=l2+size-1;
              if( h2>=R ) 
                h2=R-1;
              i=l1;
              j=l2;
              while(i<=h1 && j<=h2 )
              {
                if( access_buffer[i] <= access_buffer[j] )
                {
                  temp2_buffer[k] = req_id_buffer[i];
                  temp_buffer[k++]=access_buffer[i++];
                }
                else
                { 
                  temp2_buffer[k] = req_id_buffer[j];
                  temp_buffer[k++]=access_buffer[j++];
                }
              }
              
              while(i<=h1)
              {
                temp2_buffer[k] = req_id_buffer[i];
                temp_buffer[k++]=access_buffer[i++];

              }
              while(j<=h2)
              {
                temp2_buffer[k] = req_id_buffer[j];
                temp_buffer[k++]=access_buffer[j++];
              }
              l1=h2+1; 
            }
            for(i=l1; k<R; i++) 
            {
              temp2_buffer[k] = req_id_buffer[i];
              temp_buffer[k++]=access_buffer[i];

            }

            for(i=0;i<R;i++)
            {
              req_id_buffer[i] = temp2_buffer[i];
              access_buffer[i]=temp_buffer[i];

            }
          }
    }
    __syncthreads();
    while(id==0 && access_buffer[id]==access_buffer[id-1]);
      printf("\n%d , %d , %d\n",access_buffer[id],req_id_buffer[id],id);
  }


}

//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }

    //*********************************
    // Call the kernels here
    int temp = 0;
    for(int i=0;i<N;i++)
    {
      temp = temp + facility[i];
      facility[i] = temp;
    }
    // variable declarations...
    int *d_centre,*d_facility,*d_capacity,*d_fac_ids,*d_succ_reqs,*d_tot_reqs,*d_req_id,*d_req_cen,*d_req_fac,*d_req_start,*d_req_slots;
    
    
    // Allocate memory on GPU 
    hipMalloc( &d_req_id   , (R) * sizeof (int) );
    hipMalloc( &d_req_cen  , (R) * sizeof (int) );
    hipMalloc( &d_req_fac  , (R) * sizeof (int) );
    hipMalloc( &d_req_start, (R) * sizeof (int) );
    hipMalloc( &d_req_slots, (R) * sizeof (int) );
    
    hipMalloc( &d_centre    , N * sizeof (int)); 
    hipMalloc( &d_facility  , N * sizeof (int)); 
    hipMalloc( &d_capacity  , max_P * N  * sizeof (int));
    hipMalloc( &d_fac_ids   , max_P * N  * sizeof (int));
    hipMalloc( &d_succ_reqs , N*sizeof(int)); 
    hipMalloc( &d_tot_reqs  , N*sizeof(int)); 

    // Transferring Centre Details all at once
    hipMemcpy(d_centre   , centre   , N * sizeof (int)  , hipMemcpyHostToDevice);
    hipMemcpy(d_facility , facility , N * sizeof (int)  , hipMemcpyHostToDevice);
    hipMemcpy(d_capacity , capacity , max_P * N  * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(d_fac_ids  , fac_ids  , max_P * N  * sizeof (int), hipMemcpyHostToDevice);
    // cudaMemcpy(d_succ_reqs, succ_reqs, N*sizeof(int) , cudaMemcpyHostToDevice)
    // cudaMemcpy(d_tot_reqs , tot_reqs , N*sizeof(int) , cudaMemcpyHostToDevice)
    
    // Transferring Request in a batch of 1024
    unsigned int i = 0;
    unsigned long int req_per_iter = BLOCKSIZE * (sizeof(int));
    long int max_iter =  ceil(float(R)/BLOCKSIZE);
    printf("number of iteration required %ld \n",max_iter);

    if (max_iter>1){
      printf("Byte Transfer per cycle %ld \n",req_per_iter);
      hipMemcpy(d_req_id    , req_id     , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_cen   , req_cen    , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_fac   , req_fac    , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_start , req_start  , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_slots , req_slots  , req_per_iter, hipMemcpyHostToDevice);
    }
    else{
      hipMemcpy(d_req_id    , req_id     , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_cen   , req_cen    , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_fac   , req_fac    , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_start , req_start  , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_slots , req_slots  , (R) * sizeof (int) , hipMemcpyHostToDevice);
    }
    
    // printf("%d,%d \n",sizeof(int),sizeof(unsigned int));

    for ( i = 1 ;i<max_iter; i++){

      hipMemcpyAsync(d_req_id    + (i * req_per_iter), req_id    + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_cen   + (i * req_per_iter), req_cen   + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_fac   + (i * req_per_iter), req_fac   + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_start + (i * req_per_iter), req_start + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_slots + (i * req_per_iter), req_slots + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);

      allot_request<<<1,BLOCKSIZE>>>(d_centre,d_facility,d_capacity,d_fac_ids,d_req_id,d_req_cen,d_req_fac,d_req_start,d_req_slots,i,R);
      
      hipDeviceSynchronize();


    }

    allot_request<<<1,BLOCKSIZE>>>(d_centre,d_facility,d_capacity,d_fac_ids,d_req_id,d_req_cen,d_req_fac,d_req_start,d_req_slots,i,R);
    // mergesort<<<1,4>>>(d_req_start,50);
    hipDeviceSynchronize();
    hipMemcpy(tot_reqs , d_tot_reqs , N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(succ_reqs, d_succ_reqs, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&success  , (unsigned *)&d_succ     ,     sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&fail     , (unsigned *)&d_fail     ,     sizeof(int), hipMemcpyDeviceToHost);
    //********************************

    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}