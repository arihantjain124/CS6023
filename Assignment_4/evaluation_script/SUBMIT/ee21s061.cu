#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <numeric>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************
__global__ void init_capacity(int *capacity_per_hour,int *capacity){

  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int blx = blockIdx.x;
  int cap;
  cap = capacity[idy + (blx * gridDim.x)];
  long int id = idx + (idy * blockDim.x) + (blx * gridDim.x);
  capacity_per_hour[id] = cap;
}

__global__ void allot_request(int *facility,int *capacity,int *req_id,int *req_cen,int *req_fac,int *req_start,int *req_slots,int i,int R,int *tot_reqs,int *succ_reqs){

  unsigned int id = threadIdx.x;
  __shared__ unsigned int access_buffer[1024];
  __shared__ unsigned int req_id_buffer[1024];
  __shared__ unsigned int temp2_buffer[1024];
  __shared__ long int temp_buffer[1025];
  __shared__ volatile int size;
  size = 1;


  __syncthreads();
  unsigned int uid = req_cen[id] * 100 + req_fac[id];
  access_buffer[id] = uid;
  req_id_buffer[id] = id;
  


  __syncthreads();
  
  //Sort
  if (threadIdx.x == 0){
      int l1,l2,k,h1,h2,j;
        for(size=1; size < R; size=size*2)
        {
          l1=0;
          k=0;
          while( l1+size < R)
          {
            h1=l1+size-1;
            l2=h1+1;
            h2=l2+size-1;
            if( h2>=R ) 
              h2=R-1;
            i=l1;
            j=l2;
            while(i<=h1 && j<=h2 )
            {
              if( access_buffer[i] <= access_buffer[j] )
              {
                temp2_buffer[k] = req_id_buffer[i];
                temp_buffer[k++]=access_buffer[i++];
              }
              else
              { 
                temp2_buffer[k] = req_id_buffer[j];
                temp_buffer[k++]=access_buffer[j++];
              }
            }
            
            while(i<=h1)
            {
              temp2_buffer[k] = req_id_buffer[i];
              temp_buffer[k++]=access_buffer[i++];

            }
            while(j<=h2)
            {
              temp2_buffer[k] = req_id_buffer[j];
              temp_buffer[k++]=access_buffer[j++];
            }
            l1=h2+1; 
          }
          for(i=l1; k<R; i++) 
          {
            temp2_buffer[k] = req_id_buffer[i];
            temp_buffer[k++]=access_buffer[i];

          }

          for(i=0;i<R;i++)
          {
            req_id_buffer[i] = temp2_buffer[i];
            access_buffer[i] = temp_buffer[i];
          }
        }
  }

  __syncthreads();

  temp_buffer[id+1] = access_buffer[id];
  
  if(threadIdx.x == 0){
    temp_buffer[0] = -1;
  
  }

  temp2_buffer[id] = 0 ;

  __syncthreads();

    bool flag = temp_buffer[id+1]==temp_buffer[id];
  if(!flag)
  {
    temp2_buffer[id] = 1;
  }
  // __syncthreads();
  // if(threadIdx.x ==0)
  //   {
  //     for(int l =0;l<R;l++)
  //       printf("%d,%d,%d,%d\n\n",access_buffer[l],req_id_buffer[l],l,temp2_buffer[l]);
  //   }
    
  __syncthreads();
  if(!flag)
  {
    unsigned int curr_req;
   for(int j=id;;){
    curr_req = req_id_buffer[j];
    int start_slot = req_start[curr_req]-1;
    int end_slot = start_slot + req_slots[curr_req];

    bool pos = true;
    unsigned int base_index;
    unsigned cen = req_cen[curr_req];
    atomicAdd((unsigned *)&tot_reqs[cen],1);
    if(cen == 0){
        base_index = req_fac[curr_req] * 24;
    }
    else{
        int temp = cen-1;
        base_index = (req_fac[curr_req] + facility[temp]) * 24;
    }
    
    for(i=start_slot;i<end_slot;i++){
      if(capacity[base_index + i]==0)
        pos = false;
    }
    if(pos == true){
      for(i=start_slot;i<end_slot;i++){
        atomicAdd((int*)&capacity[base_index + i],-1);
        // capacity[base_index + i]-=1;
        }
      atomicAdd((unsigned *)&succ_reqs[cen],1);
      }
      
    // if(cen == 2 && req_fac[curr_req] == 1){
    //   printf("\ns=%d:e=%d:fac=%d:req=%d:sta=%d\n",start_slot,end_slot,req_fac[curr_req],curr_req,pos);
    //   // for(int g=0;g<24;g++){
    //   //   printf("%d ",g);
    //   // }
    //   // printf("\n");
    //   for(int g=0;g<24;g++){
    //     // if(g>10){
    //     //   printf(" ");
    //     // }
    //     printf("%d ",capacity[base_index + g]);
    //   }
    //   printf("\n");
    // }
    
    j++;
    if(temp2_buffer[j]==1 || j>R-1){
      // printf("break %d ,%d\n",j,id);
      break;
    }
    __threadfence();
    }
   }
  
  __syncthreads();
}

//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 
   

    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }

    //*********************************
    // Call the kernels here
    int temp = 0;
    for(int i=0;i<N;i++)
    {
      temp = temp + facility[i];
      facility[i] = temp;
      
    }
    // variable declarations...
    int *d_centre,*d_facility,*d_capacity,*d_fac_ids,*d_succ_reqs,*d_tot_reqs,*d_req_id,*d_req_cen,*d_req_fac,*d_req_start,*d_req_slots;
    int *capacity_per_hour;

    // Allocate memory on GPU 
    hipMalloc( &d_req_id   , (R) * sizeof (int) );
    hipMalloc( &d_req_cen  , (R) * sizeof (int) );
    hipMalloc( &d_req_fac  , (R) * sizeof (int) );
    hipMalloc( &d_req_start, (R) * sizeof (int) );
    hipMalloc( &d_req_slots, (R) * sizeof (int) );
    
    hipMalloc( &d_centre    , N * sizeof (int)); 
    hipMalloc( &d_facility  , N * sizeof (int)); 
    hipMalloc( &d_capacity  , max_P * N  * sizeof (int));
    hipMalloc( &d_fac_ids   , max_P * N  * sizeof (int));
    hipMalloc( &capacity_per_hour  , facility[N-1] * N * 24 * sizeof (int));
    hipMalloc( &d_succ_reqs , N*sizeof(int)); 
    hipMalloc( &d_tot_reqs  , N*sizeof(int)); 

    // Transferring Centre Details all at once
    hipMemcpy(d_centre   , centre   , N * sizeof (int)  , hipMemcpyHostToDevice);
    hipMemcpy(d_facility , facility , N * sizeof (int)  , hipMemcpyHostToDevice);
    hipMemcpy(d_capacity , capacity , max_P * N  * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(d_fac_ids  , fac_ids  , max_P * N  * sizeof (int), hipMemcpyHostToDevice);
    // cudaMemcpy(d_succ_reqs, succ_reqs, N*sizeof(int) , cudaMemcpyHostToDevice)
    // cudaMemcpy(d_tot_reqs , tot_reqs , N*sizeof(int) , cudaMemcpyHostToDevice)
    
    long int  gridDimx = ceil(float(N) / 40);
    long int  blockDimy = facility[N-1] % 40;
    dim3 grid3(gridDimx,1,1);
    dim3 block3(24,blockDimy,1);
    // printf("herehe %ld %ld ",gridDimx,blockDimy);
    init_capacity<<<grid3,block3>>>(capacity_per_hour,d_capacity);
    hipDeviceSynchronize();

    // Transferring Request in a batch of 1024
    unsigned int i = 0;
    unsigned long int req_per_iter = BLOCKSIZE * (sizeof(int));
    long int max_iter =  ceil(float(R)/BLOCKSIZE);
    // printf("number of iteration required %ld \n",max_iter);

    if (max_iter>1){
      // printf("Byte Transfer per cycle %ld \n",req_per_iter);
      hipMemcpy(d_req_id    , req_id     , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_cen   , req_cen    , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_fac   , req_fac    , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_start , req_start  , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpy(d_req_slots , req_slots  , req_per_iter, hipMemcpyHostToDevice);
    }
    else{
      hipMemcpy(d_req_id    , req_id     , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_cen   , req_cen    , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_fac   , req_fac    , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_start , req_start  , (R) * sizeof (int) , hipMemcpyHostToDevice);
      hipMemcpy(d_req_slots , req_slots  , (R) * sizeof (int) , hipMemcpyHostToDevice);
    }
    
    // printf("%d,%d \n",sizeof(int),sizeof(unsigned int));

    for ( i = 1 ;i<max_iter; i++){

      hipMemcpyAsync(d_req_id    + (i * req_per_iter), req_id    + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_cen   + (i * req_per_iter), req_cen   + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_fac   + (i * req_per_iter), req_fac   + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_start + (i * req_per_iter), req_start + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);
      hipMemcpyAsync(d_req_slots + (i * req_per_iter), req_slots + (i * req_per_iter) , req_per_iter, hipMemcpyHostToDevice);

      allot_request<<<1,BLOCKSIZE>>>(d_facility,capacity_per_hour,d_req_id,d_req_cen,d_req_fac,d_req_start,d_req_slots,i,R,d_tot_reqs,d_succ_reqs);
      
      hipDeviceSynchronize();


    }

    allot_request<<<1,R>>>(d_facility,capacity_per_hour,d_req_id,d_req_cen,d_req_fac,d_req_start,d_req_slots,i,R,d_tot_reqs,d_succ_reqs);
    hipDeviceSynchronize();
    // printf("Total Request %d\n",tot_reqs);
    hipMemcpy(tot_reqs , d_tot_reqs , N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(succ_reqs, d_succ_reqs, N * sizeof(int), hipMemcpyDeviceToHost);
    int total = std::accumulate(tot_reqs , tot_reqs + N , 0);
    success = std::accumulate(succ_reqs , succ_reqs + N , 0);
    fail = total - success;
    //********************************

    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}