#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void computeKernel(int p, int q, int r, int *A, int *B, 
	         int *C, int *D, int *E){
	
	__shared__ int s[512];
	// if(blockIdx.x == 0){
	long int id_e = (blockIdx.x * blockDim.x)  +   threadIdx.x;
	long int r_e = id_e / (r);
	long int c_e = id_e % (r);

	if(threadIdx.y == 0){
		for (int i =0;i<q;i++){
		E[id_e] += (A[ (r_e*q) + i] * B[ (i*r)  +  c_e ]);
		}
	}
	else{
		__syncthreads();
		s[threadIdx.x] = 0;
		for (int i =0;i<q;i++){
		s[threadIdx.x] += (C [(r_e*q) + i ] * D[ (c_e*q) + i ]) ;
	}

	}
	__syncthreads();
	E[id_e] = E[id_e] + s[threadIdx.x];
	// }
	// }

}



// function to compute the output matrix
void computE(int p, int q, int r, int *h_matrixA, int *h_matrixB, 
	         int *h_matrixC, int *h_matrixD, int *h_matrixE){
	// Device variables declarations...
	int *d_matrixA, *d_matrixB, *d_matrixC, *d_matrixD, *d_matrixE;
	
	// allocate memory...
	hipMalloc(&d_matrixA, p * q * sizeof(int));
	hipMalloc(&d_matrixB, q * r * sizeof(int));
	hipMalloc(&d_matrixC, p * q * sizeof(int));
	hipMalloc(&d_matrixD, r * q * sizeof(int));
	hipMalloc(&d_matrixE, p * r * sizeof(int));

	// copy the values...
	hipMemcpy(d_matrixA, h_matrixA, p * q * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, h_matrixB, q * r * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, h_matrixC, p * q * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixD, h_matrixD, r * q * sizeof(int), hipMemcpyHostToDevice);

	/* ****************************************************************** */
	/* Write your code here */
	/* Configure and launch kernels */
	long int gridDimx, gridDimy;
	gridDimx = ceil(float(p*r) / 1024)*2;
    dim3 grid3(gridDimx,1,1);
    dim3 block3(512,2,1);
	computeKernel<<<grid3,block3>>>(p,q,r,d_matrixA,d_matrixB,d_matrixC,d_matrixD,d_matrixE);
	hipDeviceSynchronize();
	/* ****************************************************************** */

	// copy the result back...
	hipMemcpy(h_matrixE, d_matrixE, p * r * sizeof(int), hipMemcpyDeviceToHost);

	// deallocate the memory...
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	hipFree(d_matrixD);
	hipFree(d_matrixE);
}

// function to read the input matrices from the input file
void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}

// function to write the output matrix into the output file
void writeMatrix(FILE *outputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fprintf(outputFilePtr, "%d ", matrix[i*cols+j]);
		}
		fprintf(outputFilePtr, "\n");
	}
}


/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(int *arr,  int rows,  int cols, char* filename){
    outfile.open(filename);
    for( int i = 0; i < rows; i++){
        for( int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

int main(int argc, char **argv) {
	// variable declarations
	int p, q, r;
	int *matrixA, *matrixB, *matrixC, *matrixD, *matrixE;
	struct timeval t1, t2;
	double seconds, microSeconds;

	// get file names from command line
	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	// file pointers
	FILE *inputFilePtr, *outputFilePtr;
    
    inputFilePtr = fopen(inputFileName, "r");
	if(inputFilePtr == NULL) {
	    printf("Failed to open the input file.!!\n"); 
		return 0;
	}

	// read input values
	fscanf(inputFilePtr, "%d %d %d", &p, &q, &r);

	// allocate memory and read input matrices
	matrixA = (int*) malloc(p * q * sizeof(int));
	matrixB = (int*) malloc(q * r * sizeof(int));
	matrixC = (int*) malloc(p * q * sizeof(int));
	matrixD = (int*) malloc(r * q * sizeof(int));
	readMatrix(inputFilePtr, matrixA, p, q);
	readMatrix(inputFilePtr, matrixB, q, r);
	readMatrix(inputFilePtr, matrixC, p, q);
	readMatrix(inputFilePtr, matrixD, r, q);

	// allocate memory for output matrix
	matrixE = (int*) malloc(p * r * sizeof(int));

	// call the compute function
	gettimeofday(&t1, NULL);
	computE(p, q, r, matrixA, matrixB, matrixC, matrixD, matrixE);
	hipDeviceSynchronize();
	gettimeofday(&t2, NULL);

	// print the time taken by the compute function
	seconds = t2.tv_sec - t1.tv_sec;
	microSeconds = t2.tv_usec - t1.tv_usec;
	printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);

	// store the result into the output file
	// outputFilePtr = fopen(outputFileName, "w");
	// writeMatrix(outputFilePtr, matrixE, p, r);

	// close files
	fclose(inputFilePtr);
	// fclose(outputFilePtr);

	printMatrix(matrixE, p,r,"kernel3.txt");
	// deallocate memory
	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixD);
	free(matrixE);

	return 0;
}
	
